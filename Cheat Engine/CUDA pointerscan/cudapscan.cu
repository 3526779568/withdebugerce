#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "pointerscan.cuh"
#include "pscanfileaccess.h"


#include <stdio.h>

__global__ void test()
{
  
    printf("CUDA is working\n");
    
}

int main()
{
	printf("CUDA pointerscan. WEEEEEEEE\n");
	
	test<<<1,1>>>();
	
	hipDeviceSynchronize();
	
	
	pscaninit(); 
	pointerscan(0x00201C20, 2048, 5);
	
	
	hipDeviceSynchronize();
	hipDeviceReset();


    return 0;
}
