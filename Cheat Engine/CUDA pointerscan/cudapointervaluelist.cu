#include "hip/hip_runtime.h"
#include <windows.h>
#include <stdio.h>


#include "cudapointervaluelist.cuh"
#include "bigalloc.cuh"
 


__device__ PReversePointerList Level0List=NULL;
__device__ int maxlevel;

__device__ PPointerList firstPointerValue=NULL;
__device__ PPointerList lastPointerValue=NULL;
    
__device__ PPointerList findClosestPointer(PReversePointerList addresslist, int entrynr, int level, UINT_PTR maxvalue)
/*
The pointer was not found exactly, but we are in an addresslist that has been allocated, so something is filled in at least
*/
{
	int i;
	PPointerList result=NULL;
	
	for (i=entrynr+1; i<=0xF; i++)
	{		
		if (addresslist[i].u.ReversePointerList)
		{
		  if (level==maxlevel)
		  {
			result=addresslist[i].u.PointerList;
			while ((result) && (result->PointerValue>maxvalue)) //should only run one time
			  result=result->Previous;

			if (result==NULL)
			  result=firstPointerValue;

			return result;
		  }
		  else //dig deeper
		  {
			result=findClosestPointer(addresslist[i].u.ReversePointerList, -1, level+1, maxvalue); //so it will be found by the next top scan
			if (result) 
			  return result;
		  }
		}	
	
	}


	//nothing at the top, try the bottom
	for (i=entrynr-1; i>=0; i--)
	{
	    if (addresslist[i].u.ReversePointerList)
		{
		  if (level==maxlevel) 
		  {
			result=addresslist[i].u.PointerList;
			while ((result) && (result->PointerValue>maxvalue)) //should never happen
			  result=result->Previous;

			if (result==NULL) 
			  result=firstPointerValue;

			return result;				
			
		  }
		  else //dig deeper
		  {
			result=findClosestPointer(addresslist[i].u.ReversePointerList,0x10, level+1, maxvalue); //F downto 0
			if (result) 
			  return result;
		  }
		}
	}
	return result;
	

}    

__device__ PPointerList findPointerValue(UINT_PTR startvalue, UINT_PTR *stopvalue)
/*
find a node that falls in the region of stopvalue and startvalue
*/
{
  PPointerList result=NULL;
  int level;
  PReversePointerList currentarray;
  int entrynr;  
  UINT_PTR _stopvalue; 

  _stopvalue=*stopvalue;
  currentarray=Level0List;

//  printf("findPointerValue for %x\n", (unsigned int)startvalue);
//  printf("maxlevel is %d\n", (unsigned int)maxlevel);

  for (level=0; level<=maxlevel; level++)
  {
    entrynr=((UINT64)_stopvalue >> (UINT64)(((maxlevel-level)*4))) & 0xf;
    

    if (currentarray[entrynr].u.ReversePointerList==NULL) //not found
    {
      result=findClosestPointer(currentarray, entrynr, level, _stopvalue);
      break;
    }
    else
    {
      if (level==maxlevel)
      {
        result=currentarray[entrynr].u.PointerList;
        break;
      }
    }
    currentarray=currentarray[entrynr].u.ReversePointerList;
  }

  *stopvalue=result->PointerValue;

  //clean up bad results
  
 // printf("result=%p\n", result);
 // printf("result->PointerValue=%x\n", (unsigned int)result->PointerValue);
  
 // printf("result->Next->PointerValue=%x\n", (unsigned int)result->Next->PointerValue);
  
  
  
  if (result->PointerValue<startvalue)
    result=NULL;
    
  return result;
}

 
    
    
__device__ void fillList(PReversePointerList addresslist, int level, PPointerList *prev)
/*
Fills in the linked list of the reverse pointer list
*/
{
  int i;
  if (level==maxlevel)
  {
    for (i=0; i<=0xf; i++)
    {
      if (addresslist[i].u.PointerList)
      {
        if (*prev)
          (*prev)->Next=addresslist[i].u.PointerList;
        else
          firstPointerValue=addresslist[i].u.PointerList;

        addresslist[i].u.PointerList->Previous=*prev;
        *prev=addresslist[i].u.PointerList;
      }
    }
  }
  else
  {
    for (i=0; i<=0xf; i++)
    {    
      if (addresslist[i].u.ReversePointerList)
        fillList(addresslist[i].u.ReversePointerList,level+1, prev);
    }
  }

}

__global__ void generateLinkedList(void)
{
  lastPointerValue=NULL;
  fillList(Level0List,0,&lastPointerValue); 
}

__global__ void findoraddpointervalue(unsigned char *bla, int max)
/*
Go through the data and add the pointervalue. (port from ce's pascal source, with some improvements since no dynamic loading is necesary)
*/
{
  int i;
  int loopnr;
  int pd=0;
  UINT64 pointervalue;
  DWORD pointercount;
  int level, entrynr, size;
  PReversePointerList currentarray, temp;
  PPointerList plist;

  if (Level0List==NULL) //first time init
  {
    Level0List=(PReversePointerList)malloc(16*sizeof(PReversePointerList));
    memset(Level0List, 0, 16*sizeof(PReversePointerList));
  }
 
  // printf("this one will crash\n");
  loopnr=1;
   
  while (pd<max)
  {
	 // printf("loopnr %d\n", loopnr);

	  memcpy(&pointervalue, &bla[pd], sizeof(pointervalue));
	  pd+=sizeof(pointervalue);
	  
	 // printf("pointervalue=%x\n", (unsigned int)pointervalue);
		
		   	 
	  currentarray=Level0List;
	  level=0;
	  

	  while (level<maxlevel)
	  {
	 
		//add the path if needed
		entrynr=((UINT64)pointervalue >> (UINT64)(((maxlevel-level)*4))) & 0xf;
	    
	     
		if (currentarray[entrynr].u.ReversePointerList==NULL) //allocate
		{   
		  size=16*sizeof(PReversePointerList);
		 
		  temp=(PReversePointerList)balloc(size);   
		 // memset(temp, 0, size);
	        
		  currentarray[entrynr].u.ReversePointerList=temp;      
		  
		}
		currentarray=currentarray[entrynr].u.ReversePointerList;
	    
		level++;
	  }
	  
	
	
	 
	  entrynr=((UINT64)pointervalue >> (UINT64)(((maxlevel-level)*4))) & 0xf;
	  plist=currentarray[entrynr].u.PointerList;   
	  
	  if (plist==NULL) //allocate one 
	  {
	 
		currentarray[entrynr].u.PointerList=(TPointerList*)balloc(sizeof(TPointerList));
		plist=currentarray[entrynr].u.PointerList;
		plist->PointerValue=pointervalue;

		plist->list=NULL;
		plist->pos=0;
		
	  }

	  //use the current plist
	  
	  
	  memcpy(&pointercount, &bla[pd], sizeof(pointercount));
	  pd+=sizeof(pointercount);
	  	  
   
	   
	  plist->pos=pointercount;
	  plist->list=(TPointerData *)balloc(sizeof(TPointerData)*pointercount); 
	  
	  //printf("plist->list=%p\n", plist->list);
	  
 
	  for (i=0; i<pointercount; i++)
	  {
	 
		UINT64 address;	
		memcpy(&address, &bla[pd], sizeof(address));
		pd+=sizeof(address);
		plist->list[i].address=address;
			

		
		if (bla[pd]==1)
		{
		  DWORD moduleindex;	  
		  DWORD offset;
		  pd+=1;
		  
		  memcpy(&moduleindex, &bla[pd], sizeof(moduleindex));
		  pd+=sizeof(moduleindex);
		  
		  memcpy(&offset, &bla[pd], sizeof(offset));
		  pd+=sizeof(offset);  
		  
	
		  plist->list[i].staticdata=(TStaticData *)balloc(sizeof(TStaticData));
		  plist->list[i].staticdata->moduleindex=moduleindex;
		  plist->list[i].staticdata->offset=offset;	  	  
		  
		  //printf("plist->list[i].staticdata=%p\n", plist->list[i].staticdata);
		 
		}
		else
		{
		  pd+=1;
		  plist->list[i].staticdata=NULL;
	
		}
	  } //for 

	  
	  
	 // printf("pd=%d  max=%d\n", pd, max);
	  loopnr++;
  }
 
}


void setMaxLevel(int count)
{	
	hipMemcpyToSymbol(HIP_SYMBOL(maxlevel), &count, sizeof(count));	
}