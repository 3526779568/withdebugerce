
#include <hip/hip_runtime.h>
#include <stdio.h>

//This file is used when a lot of memory needs to be allocated in small chunks
//Note: this is NOT thread safe, only let 1 thread access this code
#pragma pack 16
typedef __declspec(align(16)) struct _BigAllocs
{
	int totalsize;
	int pos;
	unsigned char *buffer;
} BigAllocs, *PBigAllocs;

__device__ BigAllocs *allocs=NULL;
__device__ int allocsPos=0;
__device__ int allocsMax=0;

__device__ void *balloc(int size)
{
	void *result=NULL;
    
	//printf("allocating %d bytes\n", size);
	
	
	if (allocs==NULL) //allocate a big amoung of memory first
	{
		printf("balloc first init\n");
		allocsMax=16;
		allocsPos=0;
		allocs=(PBigAllocs)malloc(allocsMax*sizeof(BigAllocs));		
		allocs[allocsPos].pos=0;
		allocs[allocsPos].totalsize=2*1024*1024;
		allocs[allocsPos].buffer=(unsigned char *)malloc(allocs[allocsPos].totalsize); //16MB	+1mb for each new pos	
		printf("allocs[allocsPos].buffer=%p\n", allocs[allocsPos].buffer);
		memset(allocs[allocsPos].buffer,0, allocs[allocsPos].totalsize);
	}	
	
	if (allocs[allocsPos].totalsize-allocs[allocsPos].pos<size) 
	{
		printf("balloc reinit\n");
		

		
		//a new BigAllocs object is needed
		allocsPos++;
		if (allocsPos>=allocsMax)
		{		
			PBigAllocs old=allocs;
			printf("Reallocating\n");
			
			allocs=(PBigAllocs)malloc(allocsMax*sizeof(BigAllocs)*2);
			memcpy(allocs, old, allocsMax*sizeof(BigAllocs));			
			
		    allocsMax*=2; //allocate more blocks			
		}
		
		allocs[allocsPos].pos=0;
		allocs[allocsPos].totalsize=2*1024*1024;
		allocs[allocsPos].buffer=(unsigned char *)malloc(allocs[allocsPos].totalsize);	
		memset(allocs[allocsPos].buffer,0, allocs[allocsPos].totalsize);
		
		printf("buffer=%p\n", allocs[allocsPos].buffer);
		
		
	}
	
	result=&allocs[allocsPos].buffer[allocs[allocsPos].pos];
	allocs[allocsPos].pos+=size;
	if (allocs[allocsPos].pos & 0xf) //make sure the next one is aligned
		allocs[allocsPos].pos=(allocs[allocsPos].pos + 0x10) & ~(0xf);
		

	//printf("allocs[allocsPos].pos=%x\n", allocs[allocsPos].pos);
		
	return result;	
	
}
