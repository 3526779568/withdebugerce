#define __MVS__
#define WINDOWS
#define ZLIB_DLL
#define ZLIB_WINAPI

#include <windows.h>
#include <zlib.h>

#include <stdio.h>

#include "cudapointervaluelist.cuh"

HANDLE f;
z_stream strm;



unsigned char in[16*1024];

    

int readBytes(void *buf, int count)
{
	int r;
	
	strm.avail_out=count;
	strm.next_out=(Bytef *)buf;

	while (strm.avail_out) //while there are still bytes to extract
	{
		if (strm.avail_in==0) //read a chunk
		{
			DWORD br=0;			
			if (ReadFile(f, in, 16*1024, &br, NULL)==FALSE)
			{
				printf("read error...\n");
			
			}
			strm.avail_in=br;
			
			if (strm.avail_in==0)
			{
				printf("read error...\n");
			}

			strm.next_in=in;
		}

		r = inflate(&strm, Z_NO_FLUSH);
		if (r==Z_STREAM_END) 
		{
			printf("Stream end reached");
			break;
		}

		if (r!=Z_OK)
		{
			printf("inflate error\n");
			return r;
		}
	}

	return r;
}

DWORD readBYTE()
{
	unsigned char r;
	if (readBytes(&r, 1)!=S_OK)
	{
	  printf("readBYTE failed");
	}
	
	return r;
}

DWORD readDWORD()
{
	DWORD r;
	if (readBytes(&r, 4)!=S_OK)	
	{
	  printf("readDWORD failed");
	}	
	return r;
}

UINT64 readQWORD()
{
	UINT64 r;
	if (readBytes(&r, 8)!=S_OK)
	{
		printf("readQWORD failed");	
	}
	return r;
}

int pscaninit()
{

	int ret;
	

	//char filename[]="E:\\ptr\\packed\\scanner1\\tutorial.ptr.scandata";
	char filename[]="E:\\ptr\\packed\\tst\\test.PTR.scandata";
	
	

    f=CreateFile(filename,FILE_READ_DATA, FILE_SHARE_READ|FILE_SHARE_WRITE|FILE_SHARE_DELETE, NULL, OPEN_EXISTING, FILE_FLAG_SEQUENTIAL_SCAN, NULL);
	


	if (f!=INVALID_HANDLE_VALUE)
	{
		int i;
		DWORD modulelistlength;
		DWORD maxlevel;
		UINT64 totalcount;
		
		
		unsigned char *cudainitdata=NULL;
		unsigned char *initdata=NULL;
		int datasize=0;
		int cudadatasize=0;
		DWORD bytes=0;
		int dp=0;
		

	/* allocate inflate state */
		strm.zalloc = Z_NULL;
		strm.zfree = Z_NULL;
		strm.opaque = Z_NULL;
		strm.avail_in = 0;
		strm.next_in = Z_NULL;
		ret = inflateInit(&strm);
		if (ret != Z_OK)
			return ret;

		modulelistlength=readDWORD();
		for (i=0; i<modulelistlength; i++)
		{
			char *modulename;
			DWORD modulenamelength=readDWORD();
			modulename=(char *)malloc(modulenamelength+1);
			readBytes(modulename, modulenamelength);
			modulename[modulenamelength]=0;
			printf("%s\n", modulename);

			free(modulename);			
		}

		maxlevel=readDWORD();
		totalcount=readQWORD();
		
		setMaxLevel(maxlevel);
		
		
		

		{
			UINT64 count=0;
			while (count<totalcount)
			{
			
				//calculate the length in bytes first...
			
				
				UINT64 pvalue=readQWORD();
				DWORD pointercount=readDWORD();
				TPointerData *pd=(TPointerData*)malloc(sizeof(TPointerData)*pointercount);
				
				if (pvalue==0x201be8)
				{
				  printf("it's in the list\n");
				}
				
				bytes+=12;
							
				
				
				
				for (i=0; i<pointercount; i++)
				{
					pd[i].address=readQWORD();				
				
					bytes+=9;
					
					if (readBYTE()==1)
					{
						pd[i].staticdata=(TStaticData*)malloc(sizeof(TStaticData));
						pd[i].staticdata->moduleindex=readDWORD();
						pd[i].staticdata->offset=readDWORD();
						
						bytes+=8;
					}
					else
					{
						pd[i].staticdata=NULL;
					}
				}
				
				//convert it back into a single bytelist				
				if (datasize<bytes)
				{
					datasize=bytes+128;
					if (initdata)
					{
						initdata=(unsigned char *)realloc(initdata, datasize);
					}
					else									
						initdata=(unsigned char *)malloc(datasize);
					
					
				}				
				
				*(UINT64 *)&initdata[dp]=pvalue;
				dp+=8;
				
				*(DWORD *)&initdata[dp]=pointercount;
				dp+=4;
				
				for (i=0; i<pointercount; i++)
				{				
					*(UINT64 *)&initdata[dp]=pd[i].address;
					dp+=8;
					
					*(unsigned char *)&initdata[dp]=pd[i].staticdata?1:0;
					dp+=1;
					
					if (pd[i].staticdata)
					{
						*(DWORD *)&initdata[dp]=pd[i].staticdata->moduleindex;
						dp+=4;	
						
						*(DWORD *)&initdata[dp]=pd[i].staticdata->offset;
						dp+=4;						
					}									
				}	
				
				if (bytes!=dp)			
				{
					printf("Error during initialization\n");
					exit(1);
				
				}
				
				if (bytes>4096) //enough data gathered
				{				
					//pass the bytestring to cuda
					
					hipDeviceSynchronize(); //wait till the previous findoraddpointervalue operation is done
					
					hipError_t error=hipGetLastError();
					
					if (error!=hipSuccess)
					{
						printf("Error\n");
						printf("CUDA error: %s\n", hipGetErrorString(error));  
					}

	  				
					if (cudadatasize<bytes)
					{
						if (cudainitdata)
						{
							hipFree(cudainitdata);
							cudainitdata=NULL;
						}
							
						cudadatasize=bytes+128;
						hipMalloc(&cudainitdata, cudadatasize);				
					}
									
					//send the data to the gpu
					
				//	printf("%d->%d(%d): %x - bytes=%d\n", (DWORD)pointercount, (DWORD)count, (DWORD)totalcount, (DWORD)pvalue, bytes);
				//	printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));  
					
					 
					hipError_t r=hipMemcpy(cudainitdata, initdata, bytes, hipMemcpyHostToDevice);
					findoraddpointervalue<<<1,1>>>(cudainitdata, bytes);
					bytes=0;
					dp=0;									  					

				}
				
				//cleanup the temporary pointerdata 
				for (i=0; i<pointercount; i++)
				{
				  if (pd[i].staticdata)
					free(pd[i].staticdata);
				}
				free(pd);
								
			  				
	
				count+=pointercount;
				
			}
		}
	

		if (bytes)
		{				
		    //send the last bit
			hipError_t r=hipMemcpy(cudainitdata, initdata, bytes, hipMemcpyHostToDevice);
			findoraddpointervalue<<<1,1>>>(cudainitdata, bytes);
			bytes=0;
		}
		
		
		
		
					
		if (initdata)
		  free(initdata);
		  		
		hipDeviceSynchronize();
		if (cudainitdata)
		  hipFree(cudainitdata);
		  
		hipDeviceSynchronize();  
		//build the linked list
		generateLinkedList<<<1,1>>>();
		hipDeviceSynchronize(); 
		
			
					  		 
	}

	return 0;
}

