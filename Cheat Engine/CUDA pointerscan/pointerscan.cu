#include "hip/hip_runtime.h"
/*
Pointerscan core
While the original pointerscan can run forever using recursive loops for each thread, the cuda implementation can not do that.
If a kernel does not exit within 2 seconds, it will crash. So, in order to deal with this, the recusrsive loop hase been replaced with a recursion replacement which
allows resume capability
*/
#include "hip/hip_runtime.h"

#include <windows.h>
#include <stdio.h>

#include "pscanfileaccess.h"
#include "cudapointervaluelist.cuh"

#define MAXCOMMANDLISTSIZE	2048
#pragma pack(16)

typedef __declspec(align(16)) struct _rcaller //recursion replacement
{
	UINT_PTR valueToFind;
	UINT_PTR startvalue;  
	UINT_PTR stopvalue; 
	PPointerList plist;
	int plistIndex;  //index in the plist to start off with
} rcaller, *prcaller;

typedef __declspec(align(16)) struct _continueData
{	
	int level;  //the current level	(-1 if not running)
	prcaller caller; //
	int *offsets; //temp offset list
} ContinueData, *PContinueData;

typedef __declspec(align(16)) struct _workcommand  //same as continuedata but no plist data
{
	UINT_PTR valueToFind;  
	int level;  
	int *offsets; 
} WorkCommand, *PWorkCommand;

typedef __declspec(align(16)) struct _workCommandList
{
	unsigned int lock; //not 0 when locked
	unsigned int count;
	WorkCommand list[MAXCOMMANDLISTSIZE];
} WorkCommandList, *PWorkCommandList;

__device__ WorkCommandList SavedWorkCommandList;


__device__ int didWork;


__device__ int pscan2(PContinueData cd, int structsize, int maxlevel)
/*
The pointerscanner loop

When loading take from PreviousSavedCommandList, or CurrentSavedCommandList

Thing I learned after trying to debug from 10PM to 5:30AM:  atomic functions do not work in __global__ functions
*/
{
  int timeout=4096;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int level=-1;
 
 
 //printf("blockIdx.x=%d blockDim.x=%d threadIdx.x=%d\n", blockIdx.x, blockDim.x, threadIdx.x);
//return;
 
  /*if (SavedWorkCommandList.count>MAXCOMMANDLISTSIZE)
  {
	printf("SavedWorkCommandList.count invalid at entry: %d\n", SavedWorkCommandList.count);
	return;
  }*/
  	

  //printf("pscan index=%d\n", index);

  //check if the current thread should continue from the last spot, or fetch a new work command
  if (cd[index].level==-1)
  {
    //This tread was idle. Fetch the state from the previous saved command list
    int i;/*=SavedWorkCommandList.count;
    
    if (i>MAXCOMMANDLISTSIZE)
    {
		printf("SavedWorkCommandList.count corrupted.  %d\n", i);
		
		return;
    }*/
    
    if (SavedWorkCommandList.count>0)
		i=atomicSub((unsigned int*)&SavedWorkCommandList.count, 1);
	else
		i=0;
      
   
    
    if (i>0)  //count==1 means index 0
    {  
      //something was in it
      int level;
      
       //printf("idle thread.  Fetched item %d from the SavedWorkCommandList\n", i);
      
     // printf("i>0\n");
      
      i--;
      
      /*
      if (i<0)
      {
		printf("i<0\n");
		

		return;
      }
      */
      
      level=SavedWorkCommandList.list[i].level;
      
      //debug
      /*
      if (level<0)
      {
		printf("Level(%d)<0 ERROR!\n", level);
	
		return;
      }
      
      if (level>=maxlevel)
      {
		printf("level(%d)>maxlevel(%d)\n", level, maxlevel);
		printf("i=%d\n", i);
		printf("valuetofind=%x\n", (unsigned int)SavedWorkCommandList.list[i].valueToFind);
		
		return;
      }*/
      
      cd[index].level=level;      
      
      cd[index].caller[level].valueToFind=SavedWorkCommandList.list[i].valueToFind;
      cd[index].caller[level].startvalue=SavedWorkCommandList.list[i].valueToFind-structsize;
      cd[index].caller[level].stopvalue=SavedWorkCommandList.list[i].valueToFind;      
      cd[index].caller[level].plistIndex=0;
      cd[index].caller[level].plist=NULL;
      
      if (cd[index].caller[level].valueToFind==0)
      {
		printf("cd[index].caller[level].valueToFind set to 0 by SavedWorkCommandList\n");
      }
      
     // printf("%d, %d, %p  (%d)\n", index, level, &cd[index].caller[level].valueToFind, i); 
      
      if (level)
      {
		memcpy(cd[index].offsets, SavedWorkCommandList.list[i].offsets, sizeof(int)*level); 
		cd[index].caller[level-1].valueToFind=0; //mark the previous caller as invalid (exit point)
		cd[index].caller[level-1].startvalue=0;
	  }
    /*        
      printf("Has data: SavedWorkCommandList.count=%d (i=%d)\n", SavedWorkCommandList.count, i);
      printf("cd[%d].level = %d\n", index, cd[index].level);
	  printf("cd[%d].caller[%d].valueToFind = %x\n", index, cd[index].level, (unsigned int)cd[index].caller[cd[index].level].valueToFind);      
      printf("cd[%d].caller[%d].startvalue = %x\n", index, cd[index].level, (unsigned int)cd[index].caller[cd[index].level].startvalue);
      printf("cd[%d].caller[%d].stopvalue = %x\n", index, cd[index].level, (unsigned int)cd[index].caller[cd[index].level].stopvalue);
      */
    }
    else
    {
	//  printf("No data in the list\n");	 
	  atomicExch(&SavedWorkCommandList.count, 0);
    }
      
  }
  
  
  level=cd[index].level;
  if ((level>=0) && (didWork==0))
  {
    didWork=1;
   // printf("Going to do work. Level=%d\n", level);
  }
    
  //wait till all threads are here  
  

  
  __syncthreads(); 
  
 // printf("after __syncthreads()\n");
 // printf("SavedWorkCommandList.count=%d\n", SavedWorkCommandList.count);
  
  //from now on SavedWorkCommandList will be used for output only. (First increase the count, and then use the previous value as input
  while (level>=0)
  {  
	//continue from the current state
	
	UINT_PTR valueToFind=cd[index].caller[level].valueToFind;
	UINT_PTR startvalue=cd[index].caller[level].startvalue;
	UINT_PTR stopvalue=cd[index].caller[level].stopvalue;
	PPointerList plist=cd[index].caller[level].plist;
	int plistIndex=cd[index].caller[level].plistIndex;  
	BOOL levelChanged=FALSE;
	
	
	/*if (valueToFind==0)
	{
		printf("valueTofind==0  (startvalue=%x stopvalue=%x)\n", (unsigned int)startvalue, (unsigned int)stopvalue);
		break;
	}*/
	
//	printf("level %d\n", level);
	
//	printf("%d: Processing\n", index);
//	printf("%d: valueToFind:%x\n", index, (unsigned int)valueToFind);
//	printf("%d: StartValue:%x\n", index, (unsigned int)startvalue);
//	printf("%d: StopValue: %x\n", index, (unsigned int)stopvalue);
//	printf("%d: plistIndex: %d\n", index,  plistIndex);


	
	while (stopvalue>=startvalue) 
	{	
		int currentOffset=valueToFind-stopvalue;
	
		if (plist==NULL)
		{		
			plist=findPointerValue(startvalue, &stopvalue);
			plistIndex=0;
		}
			
	//	printf("%d:plist=%p  (stopvalue=%x startvalue=%x\n", index, plist, (unsigned int)stopvalue, (unsigned int)startvalue);	
		
		if (plist)
		{
			int i;
			cd[index].offsets[level]=currentOffset;
			
			for (i=plistIndex; i<plist->pos; i++)
			{
			
				timeout--;
				if (timeout<0)
				{
				//	printf("timeout. Saving state:\n");
			/*
					if (level>=maxlevel)
					{
					   printf("level is fucked\n");
					}
					*/
								
					cd[index].level=level;
					cd[index].caller[level].valueToFind=valueToFind;
					cd[index].caller[level].startvalue=startvalue;
					cd[index].caller[level].stopvalue=stopvalue;
					cd[index].caller[level].plist=plist;
					cd[index].caller[level].plistIndex=i; //this one
					
					if (cd[index].caller[level].valueToFind==0)
					{
						printf("cd[%d].level=%d\n", index, cd[index].level);
						printf("cd[%d].caller[%d].valueToFind=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].valueToFind);
						printf("cd[%d].caller[%d].startvalue=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].startvalue);
						printf("cd[%d].caller[%d].stopvalue=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].stopvalue);
						printf("cd[%d].caller[%d].plist=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].plist);
						printf("cd[%d].caller[%d].plistIndex=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].plistIndex);
					}
					return timeout;
				}
							
				
	//			printf("%d: offset: %x\n",i, currentOffset);
				
				
				if (plist->list[i].staticdata)
				{
			//	   printf("%d: FOUND A STATIC\n", index);
				   int k;
				   
				 //  printf("plist->list[%d].address=%x\nsd=%p  offset=%x\n", i, (unsigned int)plist->list[i].address, plist->list[i].staticdata, plist->list[i].staticdata->offset);
		 
		 /*
				   printf("%x ", plist->list[i].staticdata->offset);
				   
				   for (k=level; k>=0; k--)
				   {
				     printf("- %x", cd[index].offsets[k]); 
				   }
				   
				   printf("\n");
				   */
				   
				  // timeout=timeout-1000;
				   
				}
				else
				{				
					if ((level+1)<maxlevel)
					{
						//add this path to the commandlist if possible
						int j=MAXCOMMANDLISTSIZE-10;  //Here, J is used as a countdown
						
						if (SavedWorkCommandList.count<MAXCOMMANDLISTSIZE-128)
						{
							//I could add it. Let's see if I can acuire a lock
							while (j<MAXCOMMANDLISTSIZE) //try to lock it multiple times
							{
								if (atomicExch(&SavedWorkCommandList.lock, 1)==0)
								{							
									//Acuired a lock
									
									//printf("Acquired lock after %d tries\n", MAXCOMMANDLISTSIZE-j);
									j=SavedWorkCommandList.count;
									
									if (j<MAXCOMMANDLISTSIZE) //if it's valid (there's room)
									  SavedWorkCommandList.count++;  //increase the counter
									
									atomicExch(&SavedWorkCommandList.lock, 0); //unlock		
									break;						
								}
								else
								{									
									//printf("Writer: Failure to acquire lock (%d)\n", MAXCOMMANDLISTSIZE-j);									
									j++;
								}								
							}
						}
						else
						{
							//printf("Didn't bother to add: %d\n", SavedWorkCommandList.count);
							j=MAXCOMMANDLISTSIZE;
						}
						
						//here J is used as an index into the list
						
						
			//			printf("Entering this offset. Maxlevel not reached. Got commandlist slot %d\n", j);
						
						if (j<MAXCOMMANDLISTSIZE) //we got a valid block
						{
							//add to the commandlist
			//				printf("Filling in the commandlist entry\n");
							SavedWorkCommandList.list[j].valueToFind=plist->list[i].address;
							
							if (SavedWorkCommandList.list[j].valueToFind==0)
							{
								printf("SavedWorkCommandList valueToFind was set to 0\n");								
							}
							SavedWorkCommandList.list[j].level=level+1;			
							memcpy(SavedWorkCommandList.list[j].offsets, cd[index].offsets, sizeof(int)*(level+1)); 
						
						}
						
						else
						{
						    //lock failed, or full
						    
							//do this myself
							
							
			//				printf("Entering this entry myself\n");		
							//store the current state
							
							cd[index].caller[level].valueToFind=valueToFind;
							cd[index].caller[level].startvalue=startvalue;
							cd[index].caller[level].stopvalue=stopvalue;
							cd[index].caller[level].plist=plist;
							cd[index].caller[level].plistIndex=i+1; //next one
							
							level++;						
							cd[index].caller[level].valueToFind=plist->list[i].address;
							cd[index].caller[level].startvalue=plist->list[i].address-structsize;
							cd[index].caller[level].stopvalue=plist->list[i].address;
							cd[index].caller[level].plist=NULL;
							cd[index].caller[level].plistIndex=0;
							
							if (cd[index].caller[level].valueToFind==0)
							{
								printf("cd[index].caller[level].valueToFind was set to 0\n");								
							}
														
							
			//				printf("Changed the level\n");
							levelChanged=true;
							break;											
						}
						
						
					}
				}
				
				//TIMEOUT CHECK
				
				//check if this kernel has ran long enough
				//timeout--;
				
				if (timeout<0)
				{
					//printf("timeout. Saving state:\n");
			/*
					if (level>=maxlevel)
					{
					   printf("level is fucked\n");
					}
					*/
					
					cd[index].level=level;
					cd[index].caller[level].valueToFind=valueToFind;
					cd[index].caller[level].startvalue=startvalue;
					cd[index].caller[level].stopvalue=stopvalue;
					cd[index].caller[level].plist=plist;
					cd[index].caller[level].plistIndex=i+1; //next one
					
					/*
					if (cd[index].caller[level].valueToFind==0)
					{					
						printf("cd[%d].level=%d\n", index, cd[index].level);
						printf("cd[%d].caller[%d].valueToFind=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].valueToFind);
						printf("cd[%d].caller[%d].startvalue=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].startvalue);
						printf("cd[%d].caller[%d].stopvalue=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].stopvalue);
						printf("cd[%d].caller[%d].plist=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].plist);
						printf("cd[%d].caller[%d].plistIndex=%x\n", index, cd[index].level, (unsigned int)cd[index].caller[level].plistIndex);
					}*/
					return timeout;
				}
				
				
				
			}
			
			if (levelChanged)
			{
	//			printf("Level changed. Exiting for loop\n");
			    break;
			}
			
			plistIndex=0;
			
			plist=plist->Previous;
		    if (plist)
				stopvalue=plist->PointerValue;
		    else
		    {
				break; //nothing else to be found    		
			}
			
		
		}
		else
		  break;
		  
		  
	}
	
	if (!levelChanged)
	{
		//go back if possible
		level--;
		if (level>=0)
		{
	//		printf("going up a level if possible\n");
			if (cd[index].caller[level].valueToFind)
			{
			  //it has data, select it
	//		  printf("possible\n");
			  cd[index].level=level;
			}
			else
			{
	//		  printf("Not possible. Exiting...\n");
	
			  //try to get an entry from the worklist
			  
			  //experimental:
			  {
				int i=10;
				BOOL canContinue=FALSE;
				
				if (SavedWorkCommandList.count>0) //there's something in the list
				{
					
					while (i>0) //try to acquire a lock multiple times
					{
						if (atomicExch(&SavedWorkCommandList.lock, 1)==0)
						{							
							//Acuired a lock
							
							
							
							if (SavedWorkCommandList.count) //there's still something in the list
							{			
								//printf("Acquired lock after %d tries\n", 10-i);
							
								i=SavedWorkCommandList.count-1;
								level=SavedWorkCommandList.list[i].level;			
								cd[index].level=level;					
								cd[index].caller[level].valueToFind=SavedWorkCommandList.list[i].valueToFind;								
								cd[index].caller[level].startvalue=SavedWorkCommandList.list[i].valueToFind-structsize;
								cd[index].caller[level].stopvalue=SavedWorkCommandList.list[i].valueToFind;      
      								
								cd[index].caller[level].plist=NULL;
								cd[index].caller[level].plistIndex=0;
								if (level)
								{
								    memcpy(cd[index].offsets, SavedWorkCommandList.list[i].offsets, sizeof(int)*level); 
									cd[index].caller[level-1].valueToFind=0; //mark the previous level as invalid
									cd[index].caller[level-1].startvalue=0;
								}
								
								canContinue=TRUE;
					
								SavedWorkCommandList.count--; //take it from the list
							}
							
							atomicExch(&SavedWorkCommandList.lock, 0); //unlock		
							break;						
						}
						else
						{									
							//printf("Writer: Failure to acquire lock (%d)\n", MAXCOMMANDLISTSIZE-j);									
							i--;
						}								
					}
				}
				
			    if (canContinue==FALSE)
			    {	
			      cd[index].level=-1; //end of recursive call reached, mark this thread as inactive

				  return timeout;
			    }
			  				
			  }
			  //Experimental
						
									  
		
			}		
		}
	}
//	else
//	  printf("Level just changed. Not going up\n");
  }
  
  
  if (SavedWorkCommandList.count)
	printf("waste %d\n", SavedWorkCommandList.count);
	
 // printf("Exit. Level=%d\n", level);
  cd[index].level=level;
  return timeout;
  
  
 

}

__global__ void pscan(PContinueData cd, int structsize, int maxlevel)
{
	int t;
	t=pscan2(cd, structsize, maxlevel);
//	printf("t=%d\n", t);
	
}

__global__ void initpscan(PContinueData cd, UINT_PTR address, int structsize, int maxlevel, UINT_PTR valuetofind)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	
	cd[index].caller=(prcaller)malloc(sizeof(rcaller)*maxlevel);
	cd[index].offsets=(int *)malloc(sizeof(int)*maxlevel);	
	cd[index].level=-1;	
	
	for (i=0; i<maxlevel; i++)
	{
	  cd[index].caller[i].valueToFind=0;
	  cd[index].caller[i].startvalue=0;
	  cd[index].caller[i].stopvalue=0;
	  cd[index].caller[i].plist=NULL;
	  cd[index].caller[i].plistIndex=0;
	  
	  cd[index].offsets[i]=0;
	}
	
	if (index==1)
	{
	  printf("sizeof(rcaller)=%d  (%x)\n", sizeof(rcaller), sizeof(rcaller));	
	
	  printf("\n%d : %p %p\n", index, cd[index].caller, cd[index].offsets );
	  
	  printf("c1: %p\n", &cd[index].caller[0].valueToFind); 
	  printf("c2: %p\n", &cd[index].caller[1].valueToFind); 
	  printf("c3: %p\n", &cd[index].caller[2].valueToFind);	  
	}
	
	
	
	if (cd[index].offsets==NULL)
	{
		printf("Alloc offsets failed %d\n", index);
	}
	
	if (cd[index].caller==NULL)
	{
		printf("Alloc caller failed %d\n", index);
	}
	
	if (index==0)
	{
	    int i;
		printf("initpscan %d\n", index);
		
		SavedWorkCommandList.count=0;
		
		for (i=0; i<MAXCOMMANDLISTSIZE; i++)
		{
			SavedWorkCommandList.list[i].offsets=(int *)malloc(sizeof(int)*maxlevel);
		}
		
		//setup the initial work command
		SavedWorkCommandList.list[0].level=0;
		SavedWorkCommandList.list[0].valueToFind=address;		
		SavedWorkCommandList.count=1;
		
	}


}


int pointerscan(UINT_PTR address, int structsize, int maxlevel)
{
  PContinueData cd;
  int i=0;
  BOOL done=FALSE;  
  hipError_t err;
  
  
  hipMalloc(&cd, sizeof(ContinueData)*1024);
  initpscan<<<1,1024>>>(cd, address, structsize, maxlevel+2, address);

  hipDeviceSynchronize(); 
  printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));     


  //loop till all are done
  
  while (!done)
  {	  
	  if (i%10==0)
	  {
	    int r=0;	  
		err=hipMemcpyToSymbol(HIP_SYMBOL(didWork), &r, sizeof(r));	
		if (err!=hipSuccess)  
		{
		  printf("CUDA error: %s\n", hipGetErrorString(err));  
	 	  break;
		}
		
		r=12;
		err=hipMemcpyFromSymbol(&r, HIP_SYMBOL(didWork), sizeof(r));
		if (err!=hipSuccess)  
		{
		  printf("CUDA error: %s\n", hipGetErrorString(err));  
	 	  break;
		}
		
		if (r!=0)
		{
			printf("FAIL\n");
			break;
		}		
	  }
	  
	//  printf("------------SCAN %d------------------\n", i);  
	  pscan<<<1,1024>>>(cd, structsize, 5); 
	  hipDeviceSynchronize(); 
	  
	  err=hipGetLastError();
	
	
	
	  if (err!=hipSuccess)  
	  {
		printf("CUDA error: %s\n", hipGetErrorString(err));  
		break;
	  }
	  
	  if (i%10==0)
	  {
	    int r=0;	  
		err=hipMemcpyFromSymbol(&r, HIP_SYMBOL(didWork), sizeof(r));	
		if (err!=hipSuccess)  
		{
		  printf("CUDA error: %s\n", hipGetErrorString(err));  
	 	  break;
		}
				
		if (r==0)
		  done=TRUE;
	  }	  
	  
	  i++;
	  

	  


  }
  
  //loop
  

  
   
    
  
  
  return 0;
}